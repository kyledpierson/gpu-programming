#include "hip/hip_runtime.h"
#include <fcntl.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>

#include "scatter.h"
#include "iohandler.h"
#include "Log.h"

// ============================= HELPER FUNCTIONS =============================
__constant__ float d_kernel[KERNEL_SIZE];
void copy_kernel_1D(float h_kernel[KERNEL_SIZE]) {
    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), h_kernel, KERNEL_SIZE * sizeof(float));
}

dim3 num_blocks(int x_size, int y_size, int x_threads, int y_threads) {
    // Compute the number of blocks needed for entire image
    int x_blocks = x_size / x_threads;
    int y_blocks = y_size / y_threads;

    dim3 blocks(x_blocks, y_blocks);
    return blocks;
}

// ============================= DEVICE FUNCTIONS =============================
__device__ float convolution_pixel_2D_complex(hipFloatComplex tile[BLOCKDIM_Y][BLOCKDIM_X+1], hipFloatComplex filter[KERNEL_SIZE][KERNEL_SIZE], int x, int y) {
    hipFloatComplex value = make_hipFloatComplex(0, 0);

    // Compute convolution
    for (int i = 0; i < KERNEL_SIZE; i++) {
        for (int j = 0; j < KERNEL_SIZE; j++) {
            value = hipCaddf(value, hipCmulf(tile[y-HALO_SIZE+i][x-HALO_SIZE+j], filter[KERNEL_SIZE-i-1][KERNEL_SIZE-j-1]));
        }
    }
    return hipCabsf(value);
}

__device__ float convolution_pixel_2D(float tile[BLOCKDIM_Y][BLOCKDIM_X+1], float filter[KERNEL_SIZE][KERNEL_SIZE], int x, int y) {
    float value = 0;

    // Compute convolution
    for (int i = 0; i < KERNEL_SIZE; i++) {
        for (int j = 0; j < KERNEL_SIZE; j++) {
            value += tile[y-HALO_SIZE+i][x-HALO_SIZE+j]*filter[KERNEL_SIZE-i-1][KERNEL_SIZE-j-1];
        }
    }
    return value;
}

// ============================= KERNEL FUNCTIONS =============================
__global__ void multiply(hipComplex *image, float *filter, int x_size) {
    int x = blockIdx.x * BLOCKDIM_X + threadIdx.x;
    int y = blockIdx.y * BLOCKDIM_Y + threadIdx.y;
    int offset = y*x_size + x;

    image[offset] = hipCmulf(image[offset], make_hipFloatComplex(filter[offset], 0));
}

__global__ void gaussian_convolution_2D(float *image, float *result, int x_size, int ds_x_size) {
    float gaussian_2D[7][7] = {
        {0.004922330, 0.009196123, 0.013380281, 0.015161844, 0.013380281, 0.009196123, 0.004922330},
        {0.009196123, 0.017180620, 0.024997653, 0.028326053, 0.024997653, 0.017180620, 0.009196123},
        {0.013380281, 0.024997653, 0.036371373, 0.041214164, 0.036371373, 0.024997653, 0.013380281},
        {0.015161844, 0.028326053, 0.041214164, 0.046701763, 0.041214164, 0.028326053, 0.015161844},
        {0.013380281, 0.024997653, 0.036371373, 0.041214164, 0.036371373, 0.024997653, 0.013380281},
        {0.009196123, 0.017180620, 0.024997653, 0.028326053, 0.024997653, 0.017180620, 0.009196123},
        {0.004922330, 0.009196123, 0.013380281, 0.015161844, 0.013380281, 0.009196123, 0.004922330},
    };

    // Shared memory tile for image data
    __shared__ float tile[BLOCKDIM_Y][BLOCKDIM_X+1];

    int x = threadIdx.x;
    int y = threadIdx.y;
    int x_offset = blockIdx.x*(BLOCKDIM_X-(2*HALO_SIZE))+x;
    int y_offset = blockIdx.y*(BLOCKDIM_Y-(2*HALO_SIZE))+y;

    // Load into shared memory
    tile[y][x] = image[y_offset*x_size + x_offset];
    __syncthreads();

    // Each interior thread computes output
    if (x>=HALO_SIZE && x<BLOCKDIM_X-HALO_SIZE && y>=HALO_SIZE && y<BLOCKDIM_Y-HALO_SIZE) {
        result[(y_offset/2)*ds_x_size + (x_offset/2)] = 2*convolution_pixel_2D(tile, gaussian_2D, x, y);
    }
}

__global__ void morlet_1_convolution_2D(float *image, float *result, int x_size) {
    hipFloatComplex a = make_hipFloatComplex(0.000379696, -0.000405881);
    hipFloatComplex b = make_hipFloatComplex(-0.0000489192, 0.001072378);
    hipFloatComplex c = make_hipFloatComplex(-0.001174476, -0.001103299);
    hipFloatComplex d = make_hipFloatComplex(0.001687397, 0);

    hipFloatComplex e = make_hipFloatComplex(0.004625649, -0.004944642);
    hipFloatComplex f = make_hipFloatComplex(-0.000595958, 0.01306423);
    hipFloatComplex g = make_hipFloatComplex(-0.01430805, -0.01344093);
    hipFloatComplex h = make_hipFloatComplex(0.02055671, 0);

    hipFloatComplex i = make_hipFloatComplex(0.02073072, -0.02216035);
    hipFloatComplex j = make_hipFloatComplex(-0.0026709, 0.05854983);
    hipFloatComplex k = make_hipFloatComplex(-0.06412421, -0.06023807);
    hipFloatComplex l = make_hipFloatComplex(0.09212878, 0);

    hipFloatComplex m = make_hipFloatComplex(0.03417918, -0.03653624);
    hipFloatComplex n = make_hipFloatComplex(-0.00440357, 0.09653235);
    hipFloatComplex o = make_hipFloatComplex(-0.1057229, -0.09931579);
    hipFloatComplex p = make_hipFloatComplex(0.1518947, 0);

    hipFloatComplex ac = hipConjf(a);
    hipFloatComplex bc = hipConjf(b);
    hipFloatComplex cc = hipConjf(c);

    hipFloatComplex ec = hipConjf(e);
    hipFloatComplex fc = hipConjf(f);
    hipFloatComplex gc = hipConjf(g);

    hipFloatComplex ic = hipConjf(i);
    hipFloatComplex jc = hipConjf(j);
    hipFloatComplex kc = hipConjf(k);

    hipFloatComplex mc = hipConjf(m);
    hipFloatComplex nc = hipConjf(n);
    hipFloatComplex oc = hipConjf(o);

    hipFloatComplex morlet_2D_1[7][7] = {
        {a, b, c, d, cc, bc, ac},
        {e, f, g, h, gc, fc, ec},
        {i, j, k, l, kc, jc, ic},
        {m, n, o, p, oc, nc, mc},
        {i, j, k, l, kc, jc, ic},
        {e, f, g, h, gc, fc, ec},
        {a, b, c, d, cc, bc, ac}
    };

    // Shared memory tile for image data
    __shared__ hipFloatComplex tile[BLOCKDIM_Y][BLOCKDIM_X+1];

    int x = threadIdx.x;
    int y = threadIdx.y;
    int offset = (blockIdx.y*(BLOCKDIM_Y-(2*HALO_SIZE))+y)*x_size + (blockIdx.x*(BLOCKDIM_X-(2*HALO_SIZE))+x);

    // Load into shared memory
    tile[y][x] = make_hipFloatComplex(image[offset], 0);
    __syncthreads();

    // Each interior thread computes output
    if (x>=HALO_SIZE && x<BLOCKDIM_X-HALO_SIZE && y>=HALO_SIZE && y<BLOCKDIM_Y-HALO_SIZE) {
        result[offset] = convolution_pixel_2D_complex(tile, morlet_2D_1, x, y);
    }
}

__global__ void morlet_2_convolution_2D(float *image, float *result, int x_size) {
    hipFloatComplex a = make_hipFloatComplex(0.000379696, -0.000405881);
    hipFloatComplex b = make_hipFloatComplex(-0.0000489192, 0.001072378);
    hipFloatComplex c = make_hipFloatComplex(-0.001174476, -0.001103299);
    hipFloatComplex d = make_hipFloatComplex(0.001687397, 0);

    hipFloatComplex e = make_hipFloatComplex(0.004625649, -0.004944642);
    hipFloatComplex f = make_hipFloatComplex(-0.000595958, 0.01306423);
    hipFloatComplex g = make_hipFloatComplex(-0.01430805, -0.01344093);
    hipFloatComplex h = make_hipFloatComplex(0.02055671, 0);

    hipFloatComplex i = make_hipFloatComplex(0.02073072, -0.02216035);
    hipFloatComplex j = make_hipFloatComplex(-0.0026709, 0.05854983);
    hipFloatComplex k = make_hipFloatComplex(-0.06412421, -0.06023807);
    hipFloatComplex l = make_hipFloatComplex(0.09212878, 0);

    hipFloatComplex m = make_hipFloatComplex(0.03417918, -0.03653624);
    hipFloatComplex n = make_hipFloatComplex(-0.00440357, 0.09653235);
    hipFloatComplex o = make_hipFloatComplex(-0.1057229, -0.09931579);
    hipFloatComplex p = make_hipFloatComplex(0.1518947, 0);

    hipFloatComplex ac = hipConjf(a);
    hipFloatComplex bc = hipConjf(b);
    hipFloatComplex cc = hipConjf(c);

    hipFloatComplex ec = hipConjf(e);
    hipFloatComplex fc = hipConjf(f);
    hipFloatComplex gc = hipConjf(g);

    hipFloatComplex ic = hipConjf(i);
    hipFloatComplex jc = hipConjf(j);
    hipFloatComplex kc = hipConjf(k);

    hipFloatComplex mc = hipConjf(m);
    hipFloatComplex nc = hipConjf(n);
    hipFloatComplex oc = hipConjf(o);

    hipFloatComplex morlet_2D_2[7][7] = {
        {a, e, i, m, i, e, a},
        {b, f, j, n, j, f, b},
        {c, g, k, o, k, g, c},
        {d, h, l, p, l, h, d},
        {cc, gc, kc, oc, kc, gc, cc},
        {bc, fc, jc, nc, jc, fc, bc},
        {ac, ec, ic, mc, ic, ec, ac}
    };

    // Shared memory tile for image data
    __shared__ hipFloatComplex tile[BLOCKDIM_Y][BLOCKDIM_X+1];

    int x = threadIdx.x;
    int y = threadIdx.y;
    int offset = (blockIdx.y*(BLOCKDIM_Y-(2*HALO_SIZE))+y)*x_size + (blockIdx.x*(BLOCKDIM_X-(2*HALO_SIZE))+x);

    // Load into shared memory
    tile[y][x] = make_hipFloatComplex(image[offset], 0);
    __syncthreads();

    // Each interior thread computes output
    if (x>=HALO_SIZE && x<BLOCKDIM_X-HALO_SIZE && y>=HALO_SIZE && y<BLOCKDIM_Y-HALO_SIZE) {
        result[offset] = convolution_pixel_2D_complex(tile, morlet_2D_2, x, y);
    }
}

__global__ void gaussian_convolution_row(float *image, float *result, int x_size, int y_size) {
    __shared__ float tile[BLOCKDIM_Y][(RESULT_STEPS + 2*HALO_STEPS) * BLOCKDIM_X];

    // Offset to the left halo edge
    const int x_start = (blockIdx.x * RESULT_STEPS - HALO_STEPS) * BLOCKDIM_X + threadIdx.x;
    const int y_start = blockIdx.y * BLOCKDIM_Y + threadIdx.y;

    image += y_start * x_size + x_start;
    result += y_start * x_size + x_start;

#pragma unroll
    // Load left halo
    for (int i = 0; i < HALO_STEPS; i++) {
        tile[threadIdx.y][threadIdx.x + i * BLOCKDIM_X] = (x_start >= -i * BLOCKDIM_X) ? image[i * BLOCKDIM_X] : 0;
    }

#pragma unroll
    // Load main data
    for (int i = HALO_STEPS; i < HALO_STEPS + RESULT_STEPS; i++) {
        tile[threadIdx.y][threadIdx.x + i * BLOCKDIM_X] = image[i * BLOCKDIM_X];
    }

#pragma unroll
    // Load right halo
    for (int i = HALO_STEPS + RESULT_STEPS; i < HALO_STEPS + RESULT_STEPS + HALO_STEPS; i++) {
        tile[threadIdx.y][threadIdx.x + i * BLOCKDIM_X] = (x_size - x_start > i * BLOCKDIM_X) ? image[i * BLOCKDIM_X] : 0;
    }

    __syncthreads();

#pragma unroll
    // Compute results
    for (int i = HALO_STEPS; i < HALO_STEPS + RESULT_STEPS; i++) {
        float sum = 0;

#pragma unroll
        for (int j = -HALO_SIZE; j <= HALO_SIZE; j++) {
            sum += d_kernel[HALO_SIZE - j] * tile[threadIdx.y][threadIdx.x + i * BLOCKDIM_X + j];
        }
        result[i * BLOCKDIM_X] = sum;
    }
}

__global__ void gaussian_convolution_col(float *image, float *result, int x_size, int y_size) {
    __shared__ float tile[BLOCKDIM_X][(RESULT_STEPS + 2*HALO_STEPS) * BLOCKDIM_Y + 1];

    // Offset to the upper halo edge
    const int x_start = blockIdx.x * BLOCKDIM_X + threadIdx.x;
    const int y_start = (blockIdx.y * RESULT_STEPS - HALO_STEPS) * BLOCKDIM_Y + threadIdx.y;
    image += y_start * x_size + x_start;
    result += y_start * x_size + x_start;

#pragma unroll
    //Upper halo
    for (int i = 0; i < HALO_STEPS; i++) {
        tile[threadIdx.x][threadIdx.y + i * BLOCKDIM_Y] = (y_start >= -i * BLOCKDIM_Y) ? image[i * BLOCKDIM_Y * x_size] : 0;
    }

#pragma unroll
    //Main data
    for (int i = HALO_STEPS; i < HALO_STEPS + RESULT_STEPS; i++) {
        tile[threadIdx.x][threadIdx.y + i * BLOCKDIM_Y] = image[i * BLOCKDIM_Y * x_size];
    }

#pragma unroll
    //Lower halo
    for (int i = HALO_STEPS + RESULT_STEPS; i < HALO_STEPS + RESULT_STEPS + HALO_STEPS; i++) {
        tile[threadIdx.x][threadIdx.y + i * BLOCKDIM_Y] = (y_size - y_start > i * BLOCKDIM_Y) ? image[i * BLOCKDIM_Y * x_size] : 0;
    }

    __syncthreads();

#pragma unroll
    //Compute results
    for (int i = HALO_STEPS; i < HALO_STEPS + RESULT_STEPS; i++) {
        float sum = 0;

#pragma unroll
        for (int j = -HALO_SIZE; j <= HALO_SIZE; j++) {
            sum += d_kernel[HALO_SIZE - j] * tile[threadIdx.x][threadIdx.y + i * BLOCKDIM_Y + j];
        }
        result[i * BLOCKDIM_Y * x_size] = sum;
    }
}

__global__ void downsample(float *image, float *ds_image, int x_size, int ds_x_size) {
    int x_offset = blockIdx.x*BLOCKDIM_X+threadIdx.x;
    int y_offset = blockIdx.y*BLOCKDIM_Y+threadIdx.y;

    // Save every other pixel in downsampled image
    ds_image[y_offset*ds_x_size + x_offset] = 2*image[2*(y_offset*x_size + x_offset)];
}

// ============================================================================
// ============================================================================
// ============================================================================
void gaussian_convolution_1D(Job* job, hipStream_t stream, float* d_image, float* d_result, int x_size, int y_size, int bytes, int ds_x_size, int ds_y_size, int ds_bytes) {
    dim3 blocks_row(x_size / (RESULT_STEPS * BLOCKDIM_X), y_size / BLOCKDIM_Y);
    dim3 blocks_col(x_size / BLOCKDIM_X, y_size / (RESULT_STEPS * BLOCKDIM_Y));
    dim3 ds_blocks = num_blocks(ds_x_size, ds_y_size, BLOCKDIM_X, BLOCKDIM_Y);
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);

    float *d_buffer_row, *d_buffer_col;
    hipMalloc((float**) &d_buffer_row, bytes);
    hipMalloc((float**) &d_buffer_col, bytes);
    hipMemset(d_buffer_row, 0, bytes);
    hipMemset(d_buffer_col, 0, bytes);

    gaussian_convolution_row<<<blocks_row, threads,0,stream>>>(d_image, d_buffer_row, x_size, y_size);
    gaussian_convolution_col<<<blocks_col, threads,0,stream>>>(d_buffer_row, d_buffer_col, x_size, y_size);
    downsample<<<ds_blocks, threads,0,stream>>>(d_buffer_col, d_result, x_size, ds_x_size);

    //Might want to re-work this
    job->addFree(d_buffer_row,true);
    job->addFree(d_buffer_col,true);
}

void initConsts() {
    float gaussian_1D[7] = {0.071303, 0.131514, 0.189879, 0.214607, 0.189879, 0.131514, 0.071303};
    copy_kernel_1D(gaussian_1D);
}

void scatter(float *image, JobScheduler* scheduler, const std::string& outputFile,
             int x_size, int y_size, int bytes,
             int ds_x_size_1, int ds_y_size_1, int ds_bytes_1,
             int ds_x_size_2, int ds_y_size_2, int ds_bytes_2, bool fourier, bool separable) {

    int x_active = BLOCKDIM_X-(2*HALO_SIZE);
    int y_active = BLOCKDIM_Y-(2*HALO_SIZE);

    uint64_t totalRequiredMemory = 0;
    totalRequiredMemory += (ds_bytes_1*5) + (ds_bytes_2*5) + (bytes*3);
    Job* job = scheduler->addJob();
    auto lambda = [=] (hipStream_t stream) {
        //printf("Executing job lambda...\n");

        dim3 blocks = num_blocks(x_size, y_size, x_active, y_active);
        dim3 ds_blocks = num_blocks(ds_x_size_1, ds_y_size_1, x_active, y_active);
        dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);

        // Variables
        float *d_image, *lp_1, *lp_2, *lp_3, *lp_4, *lp_5, *lp_6, *lp_7, *lp_8, *hp_1, *hp_2, *hp_3, *hp_4;

        // ----------------------------------------------------------------------------------------------------
        hipMalloc((float**) &d_image, bytes);
        hipMemcpy(d_image, image, bytes, hipMemcpyHostToDevice);

        hipMalloc((float**) &lp_1, ds_bytes_1);
        hipMalloc((float**) &hp_1, bytes);
        hipMalloc((float**) &hp_2, bytes);
        hipMemset(lp_1, 0, ds_bytes_1);
        hipMemset(hp_1, 0, bytes);
        hipMemset(hp_2, 0, bytes);

        if (fourier) {
            hipfftHandle plan_r2c, plan_c2r;
            hipfftPlan2d(&plan_r2c, y_size, x_size, HIPFFT_R2C);
            hipfftPlan2d(&plan_c2r, y_size, x_size, HIPFFT_C2R);

            // Create complex image on device
            hipComplex *c_image, *dc_image;
            int c_bytes = x_size * y_size * sizeof(hipComplex);
            hipMalloc((hipComplex**) &c_image, c_bytes);
            hipMalloc((hipComplex**) &dc_image, c_bytes);

            // Convert the image to the Fourier domain
            hipfftExecR2C(plan_r2c, d_image, c_image);

            // Read the gaussian filter (Fourier domain) ==========================================
            read_filter("gaussian_480_640.txt", image);
            hipMemcpy(d_image, image, bytes, hipMemcpyHostToDevice);

            // Perform multiplication in the Fourier domain
            hipMemcpy(dc_image, c_image, bytes, hipMemcpyDeviceToDevice);
            multiply<<<blocks, threads, 0, stream>>>(dc_image, d_image, x_size);

            // Convert the image back to the spatial domain and downsample
            hipfftExecC2R(plan_c2r, dc_image, d_image);
            downsample<<<ds_blocks, threads>>>(d_image, lp_1, x_size, ds_x_size_1);

            // Read the morlet 1 filter (Fourier domain) ==========================================
            read_filter("morlet_1_480_640.txt", image);
            hipMemcpy(d_image, image, bytes, hipMemcpyHostToDevice);

            // Perform multiplication in the Fourier domain
            hipMemcpy(dc_image, c_image, bytes, hipMemcpyDeviceToDevice);
            multiply<<<blocks, threads, 0, stream>>>(dc_image, d_image, x_size);

            // Convert the image back to the spatial domain and downsample
            hipfftExecC2R(plan_c2r, dc_image, hp_1);

            // Read the morlet 2 filter (Fourier domain) ==========================================
            read_filter("morlet_2_480_640.txt", image);
            hipMemcpy(d_image, image, bytes, hipMemcpyHostToDevice);

            // Perform multiplication in the Fourier domain
            hipMemcpy(dc_image, c_image, bytes, hipMemcpyDeviceToDevice);
            multiply<<<blocks, threads, 0, stream>>>(dc_image, d_image, x_size);

            // Convert the image back to the spatial domain and downsample
            hipfftExecC2R(plan_c2r, dc_image, hp_2);

            // Free memory
            hipFree(c_image);
            hipFree(dc_image);
        } else {
            if (separable) {
                gaussian_convolution_1D(job,stream,d_image, lp_1, x_size, y_size, bytes, ds_x_size_1, ds_y_size_1, ds_bytes_1);
            } else {
                gaussian_convolution_2D<<<blocks, threads,0,stream>>>(d_image, lp_1, x_size, ds_x_size_1);
            }
            morlet_1_convolution_2D<<<blocks, threads,0,stream>>>(d_image, hp_1, x_size);
            morlet_2_convolution_2D<<<blocks, threads,0,stream>>>(d_image, hp_2, x_size);
        }
        free(image);
        hipFree(d_image);

        // ----------------------------------------------------------------------------------------------------
        hipMalloc((float**) &lp_3, ds_bytes_1);
        hipMemset(lp_3, 0, ds_bytes_1);
        if (separable) {
            gaussian_convolution_1D(job,stream,hp_1, lp_3, x_size, y_size, bytes, ds_x_size_1, ds_y_size_1, ds_bytes_1);
        } else {
            gaussian_convolution_2D<<<blocks, threads,0,stream>>>(hp_1, lp_3, x_size, ds_x_size_1);
        }
        hipFree(hp_1);

        // ----------------------------------------------------------------------------------------------------
        hipMalloc((float**) &lp_5, ds_bytes_1);
        hipMemset(lp_5, 0, ds_bytes_1);
        if (separable) {
            gaussian_convolution_1D(job,stream,hp_2, lp_5, x_size, y_size, bytes, ds_x_size_1, ds_y_size_1, ds_bytes_1);
        } else {
            gaussian_convolution_2D<<<blocks, threads,0,stream>>>(hp_2, lp_5, x_size, ds_x_size_1);
        }
        hipFree(hp_2);

        // ----------------------------------------------------------------------------------------------------
        hipMalloc((float**) &lp_2, ds_bytes_2);
        hipMalloc((float**) &hp_3, ds_bytes_1);
        hipMalloc((float**) &hp_4, ds_bytes_1);
        hipMemset(lp_2, 0, ds_bytes_2);
        hipMemset(hp_3, 0, ds_bytes_1);
        hipMemset(hp_4, 0, ds_bytes_1);
        if (separable) {
            gaussian_convolution_1D(job,stream,lp_1, lp_2, ds_x_size_1, ds_y_size_1, ds_bytes_1, ds_x_size_2, ds_y_size_2, ds_bytes_2);
        } else {
            gaussian_convolution_2D<<<ds_blocks, threads,0,stream>>>(lp_1, lp_2, ds_x_size_1, ds_x_size_2);
        }
        morlet_1_convolution_2D<<<ds_blocks, threads,0,stream>>>(lp_1, hp_3, ds_x_size_1);
        morlet_2_convolution_2D<<<ds_blocks, threads,0,stream>>>(lp_1, hp_4, ds_x_size_1);
        hipFree(lp_1);

        // ----------------------------------------------------------------------------------------------------
        hipMalloc((float**) &lp_4, ds_bytes_2);
        hipMemset(lp_4, 0, ds_bytes_2);
        if (separable) {
            gaussian_convolution_1D(job,stream,lp_3, lp_4, ds_x_size_1, ds_y_size_1, ds_bytes_1, ds_x_size_2, ds_y_size_2, ds_bytes_2);
        } else {
            gaussian_convolution_2D<<<ds_blocks, threads,0,stream>>>(lp_3, lp_4, ds_x_size_1, ds_x_size_2);
        }
        hipFree(lp_3);

        // ----------------------------------------------------------------------------------------------------
        hipMalloc((float**) &lp_6, ds_bytes_2);
        hipMemset(lp_6, 0, ds_bytes_2);
        if (separable) {
            gaussian_convolution_1D(job,stream,lp_5, lp_6, ds_x_size_1, ds_y_size_1, ds_bytes_1, ds_x_size_2, ds_y_size_2, ds_bytes_2);
        } else {
            gaussian_convolution_2D<<<ds_blocks, threads,0,stream>>>(lp_5, lp_6, ds_x_size_1, ds_x_size_2);
        }
        hipFree(lp_5);

        // ----------------------------------------------------------------------------------------------------
        hipMalloc((float**) &lp_7, ds_bytes_2);
        hipMemset(lp_7, 0, ds_bytes_2);
        if (separable) {
            gaussian_convolution_1D(job,stream,hp_3, lp_7, ds_x_size_1, ds_y_size_1, ds_bytes_1, ds_x_size_2, ds_y_size_2, ds_bytes_2);
        } else {
            gaussian_convolution_2D<<<ds_blocks, threads,0,stream>>>(hp_3, lp_7, ds_x_size_1, ds_x_size_2);
        }
        hipFree(hp_3);

        // ----------------------------------------------------------------------------------------------------
        hipMalloc((float**) &lp_8, ds_bytes_2);
        hipMemset(lp_8, 0, ds_bytes_2);
        if (separable) {
            gaussian_convolution_1D(job,stream,hp_4, lp_8, ds_x_size_1, ds_y_size_1, ds_bytes_1, ds_x_size_2, ds_y_size_2, ds_bytes_2);
        } else {
            gaussian_convolution_2D<<<ds_blocks, threads,0,stream>>>(hp_4, lp_8, ds_x_size_1, ds_x_size_2);
        }
        hipFree(hp_4);

        // ========================================================================
        job->registerCleanup([=] () {
            //printf("Executing cleanup\n");
            int *iresult = (int*) mem_check(malloc(ds_bytes_2*5));
            float *result = (float*) mem_check(malloc(ds_bytes_2*5));
            int offset = ds_x_size_2*ds_y_size_2;

            hipMemcpy(result, lp_2, ds_bytes_2, hipMemcpyDeviceToHost);
            hipMemcpy(result+offset, lp_4, ds_bytes_2, hipMemcpyDeviceToHost);
            hipMemcpy(result+2*offset, lp_6, ds_bytes_2, hipMemcpyDeviceToHost);
            hipMemcpy(result+3*offset, lp_7, ds_bytes_2, hipMemcpyDeviceToHost);
            hipMemcpy(result+4*offset, lp_8, ds_bytes_2, hipMemcpyDeviceToHost);

            // Find the max for each image
            float maxval_1 = 0;
            float maxval_2 = 0;
            float maxval_3 = 0;
            float maxval_4 = 0;
            float maxval_5 = 0;
            for(int i = 0; i < offset*5; i++) {
                if (i/offset == 0 && result[i] > maxval_1) {
                    maxval_1 = result[i];
                } else if (i/offset == 1 && result[i] > maxval_2) {
                    maxval_2 = result[i];
                } else if (i/offset == 2 && result[i] > maxval_3) {
                    maxval_3 = result[i];
                } else if (i/offset == 3 && result[i] > maxval_4) {
                    maxval_4 = result[i];
                } else if (i/offset == 4 && result[i] > maxval_5) {
                    maxval_5 = result[i];
                }
            }

            // Re-normalize each image to a scale of 0-255
            for(int i = 0; i < offset*5; i++) {
                if (i/offset == 0) {
                    iresult[i] = (result[i] / maxval_1) * 255;
                } else if (i/offset == 1) {
                    iresult[i] = (result[i] / maxval_2) * 255;
                } else if (i/offset == 2) {
                    iresult[i] = (result[i] / maxval_3) * 255;
                } else if (i/offset == 3) {
                    iresult[i] = (result[i] / maxval_4) * 255;
                } else if (i/offset == 4) {
                    iresult[i] = (result[i] / maxval_5) * 255;
                }
            }
            LOG_DEBUG(std::string("Writing to output file: ") + outputFile);
            write_ppm((char*)outputFile.c_str(), ds_x_size_2, ds_y_size_2*5, 255, iresult);

            job->FreeMemory();

            // Free memory
            free(result);
            free(iresult);
            hipFree(lp_2);
            hipFree(lp_4);
            hipFree(lp_6);
            hipFree(lp_7);
            hipFree(lp_8);
            LOG_DEBUG("Cleanup complete");
        });
        job->setDone(); // do this when you're ready to call your cleanup
        hipStreamAddCallback(stream,&Job::cudaCb,(void*)job,0);
    };

    job->addStage(lambda,totalRequiredMemory,bytes);
    job->queue();
}


