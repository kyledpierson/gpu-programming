#include "hip/hip_runtime.h"
#include <complex.h>
#include "hip/hip_runtime.h"
#include <fcntl.h>
#include <stdlib.h>
#include <stdio.h>
#include "string.h"
#include <unistd.h>

#include "scatter.h"

// ============================= HELPER FUNCTIONS =============================
dim3 num_blocks(int x_size, int y_size, int x_threads, int y_threads) {
    // Compute the number of blocks needed for entire image
    if (x_size % x_threads) {
        x_size = x_size/x_threads*x_threads + x_threads;
    }
    if (y_size % y_threads) {
        y_size = y_size/y_threads*y_threads + y_threads;
    }
    int x_blocks = x_size / x_threads;
    int y_blocks = y_size / y_threads;

    dim3 blocks(x_blocks, y_blocks);
    return blocks;
}

__global__ void downsample(int *image, int *ds_image, int x_size, int ds_x_size) {
    int x_offset = blockIdx.x*BLOCKDIM_X+threadIdx.x;
    int y_offset = blockIdx.y*BLOCKDIM_Y+threadIdx.y;

    // Save every other pixel in downsampled image
    ds_image[y_offset*ds_x_size + x_offset] = image[2*(y_offset*x_size + x_offset)];
}

// ============================= KERNEL FUNCTIONS =============================
__device__ float convolution_pixel_2D(unsigned int tile[BLOCKDIM_Y][BLOCKDIM_X+1], float filter[KERNEL_SIZE][KERNEL_SIZE], int x, int y) {
    float value = 0;

    // Compute convolution
    for (int i = 0; i < KERNEL_SIZE; i++) {
        for (int j = 0; j < KERNEL_SIZE; j++) {
            value += tile[y-HALO_SIZE+i][x-HALO_SIZE+j]*filter[KERNEL_SIZE-i-1][KERNEL_SIZE-j-1];
        }
    }

    return value;
}

__global__ void convolution_2D(unsigned int *image, int *result, int x_size, int y_size) {
    float gaussian_2D[7][7] = {
        {0.00000019425474,  0.000096568274, 0.00010062644,  0.00021978836,  0.00010062644,  0.000096568274, 0.00000019425474},
        {0.0000096568274,	0.00048006195,	0.0050023603,	0.010926159,	0.0050023603,	0.00048006195,	0.0000096568274},
        {0.00010062644,     0.0050023603,	0.052125789,    0.11385319,	    0.052125789,	0.0050023603,	0.00010062644},
        {0.00021978836,     0.010926159,	0.11385319,	    0.24867822,	    0.11385319,	    0.010926159,    0.00021978836},
        {0.00010062644,     0.0050023603,	0.052125789,    0.11385319,	    0.052125789,	0.0050023603,	0.00010062644},
        {0.0000096568274,	0.00048006195,	0.0050023603,	0.010926159,	0.0050023603,	0.00048006195,	0.0000096568274},
        {0.00000019425474,  0.000096568274, 0.00010062644,  0.00021978836,  0.00010062644,  0.000096568274, 0.00000019425474},
    };

    // Shared memory tile for image data
    __shared__ unsigned int tile[BLOCKDIM_Y][BLOCKDIM_X+1];

    int x = threadIdx.x;
    int y = threadIdx.y;
    int offset = (blockIdx.y*(blockDim.y-(2*HALO_SIZE))+y)*x_size + (blockIdx.x*(blockDim.x-(2*HALO_SIZE))+x);

    // Load into shared memory
    tile[y][x] = image[offset];
    __syncthreads();

    // Each interior thread computes output
    if (x>=HALO_SIZE && x<blockDim.x-HALO_SIZE && y>=HALO_SIZE && y<blockDim.y-HALO_SIZE) {
        result[offset] = convolution_pixel_2D(tile, gaussian_2D, x, y);
    }
}

// ============================================================================
// ============================================================================
// ============================================================================
void scatter(unsigned int *image, int *result, int x_size, int y_size, int bytes, int ds_x_size, int ds_y_size, int ds_bytes) {
    // ====================== VARIABLES FOR CONVOLUTION =======================
    int x_active = BLOCKDIM_X-(2*HALO_SIZE);
    int y_active = BLOCKDIM_Y-(2*HALO_SIZE);

    dim3 blocks = num_blocks(x_size, y_size, x_active, y_active);
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);

    // Allocate memory
    unsigned int *d_image;
    hipMalloc((unsigned int**) &d_image, bytes);
    hipMemcpy(d_image, image, bytes, hipMemcpyHostToDevice);

    int *d_result;
    hipMalloc((int**) &d_result, bytes);
    hipMemset(d_result, 0, bytes);

    // ====================== VARIABLES FOR DOWNSAMPLING ======================
    dim3 ds_blocks = num_blocks(ds_x_size, ds_y_size, BLOCKDIM_X, BLOCKDIM_Y);

    // Allocate memory
    int *ds_result;
    hipMalloc((int**) &ds_result, ds_bytes);
    hipMemset(ds_result, 0, ds_bytes);

    // ===================== CONVOLUTION AND DOWNSAMPLING =====================
    float elapsed_time;
    hipEvent_t start,stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Convolve and downsample
    convolution_2D<<<blocks, threads>>>(d_image, d_result, x_size, y_size);
    downsample<<<ds_blocks, threads>>>(d_result, ds_result, x_size, ds_x_size);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time,start, stop);

    // Copy the result
    hipMemcpy(result, ds_result, ds_bytes, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_image);
    hipFree(d_result);
    hipFree(ds_result);

    fprintf(stderr, "TIME: %4.4f\n", elapsed_time);
}

